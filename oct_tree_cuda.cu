#include "hip/hip_runtime.h"
//
// Created by alexzhou on 4/28/2022.
// 
// nvcc -arch sm_35 oct_tree_cuda.cu -o oct_tree_cuda
// ./oct_tree_cuda

#include "oct_tree.h"
#include "rand_gen.h"
#include <malloc.h>
#include <math.h>
#include <string.h>

#include <cstdio>
#include <cstdlib>

#define CUDA_SAFE_CALL(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
	if (code != hipSuccess) 
	{
		fprintf(stderr,"CUDA_SAFE_CALL: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

//region idx tree traversal

struct Leaf* getLeaf(struct OctTree* tree, node_idx_t node_idx, child_pos_idx_t leaf_idx) {
    dbgAssert(getNode(tree, node_idx)->contentType == CT_LEAVES);
    dbgAssert(getNode(tree, node_idx)->leaves[leaf_idx] < tree->leaf_count);
    return &tree->leaves[getNode(tree, node_idx)->leaves[leaf_idx]];
}

struct OctNode* getNode(struct OctTree* tree, node_idx_t node_idx) {
    depth_t depth_idx = get_depth_for_idx(node_idx);
    dbgAssert(depth_idx < tree->depth_count);
    node_idx_t node_idx_in_depth = node_idx - idx_start_for_depth(depth_idx);
    dbgAssert(((node_idx_in_depth+1) * sizeof(struct OctNode))  <= malloc_usable_size(tree->depth[depth_idx]));
    return &tree->depth[depth_idx][node_idx_in_depth];
}

node_idx_t depth_size(depth_t depth) {
    node_idx_t idx = 1;
    for(depth_t i = 0; i < depth; i++) {
        idx *= NODE_CHILD_COUNT;
    }
    return idx;
}

node_idx_t array_size_for_depth(depth_t depth) {
    if(depth == 0) return depth_size(depth);
    return depth_size(depth) + array_size_for_depth(depth-1);
}

node_idx_t idx_start_for_depth(depth_t depth) {
    if(depth == 0) return 0;
    return array_size_for_depth(depth-1);
}

depth_t get_depth_for_idx(node_idx_t idx) {
    depth_t depth = 1;
    node_idx_t nextIdx = idx_start_for_depth(depth);
    while(idx >= nextIdx) {
        depth++;
        nextIdx = idx_start_for_depth(depth);
    }
    return depth-1;
}

node_idx_t get_node_children(node_idx_t idx) {
    depth_t depth = get_depth_for_idx(idx);
    node_idx_t depth_start = idx_start_for_depth(depth);
    node_idx_t next_depth_start = idx_start_for_depth(depth+1);
    return next_depth_start + (idx-depth_start)*NODE_CHILD_COUNT;
}

node_idx_t get_node_parent(node_idx_t idx) {
    depth_t depth = get_depth_for_idx(idx);
    node_idx_t pos = (idx - idx_start_for_depth(depth)) / NODE_CHILD_COUNT;
    return idx_start_for_depth(depth-1) + pos;
}

void set_tree_depth(struct OctTree* tree, depth_t depth_count) {
    dbgAssert(tree->depth_count < depth_count);
    dbgAssert(depth_count <= DEPTH_LIMIT);

    depth_t old_depth = tree->depth_count;
    struct OctNode** old_depth_ptr = tree->depth;

    tree->depth = calloc(depth_count, sizeof(struct OctNode*));
    tree->depth_count = depth_count;
    memset(tree->depth, 0, depth_count*sizeof(struct OctNode*));

    if(old_depth_ptr != nullptr) {
        memcpy(tree->depth, old_depth_ptr, old_depth * sizeof(struct OctNode*));
        free(old_depth_ptr);
    }

    for(depth_t i = old_depth; i < tree->depth_count; i++) {
        tree->depth[i] = calloc(depth_size(i), sizeof(struct OctNode));
        memset(tree->depth[i], 0, depth_size(i) * sizeof(struct OctNode));
        dbgAssert(depth_size(i) * sizeof(struct OctNode*) <= malloc_usable_size(tree->depth[i]));
    }
}

void walk_tree(struct OctTree* tree, node_idx_t idx, struct Extents* ext, bool (*process_callback)(struct OctTree* tree, node_idx_t idx, struct Extents* ext, void* callbackArg), void* callback_arg) {
    if(getNode(tree, idx)->contentType == CT_EMPTY) return;

    if(!process_callback(tree, idx, ext, callback_arg)) return;

    if(getNode(tree, idx)->contentType == CT_NODES) {
        node_idx_t children_idx = get_node_children(idx);

        for(int i = 0; i < NODE_CHILD_COUNT; i++) {
            node_idx_t child_idx = children_idx + i;
            struct Extents childExt = *ext;
            update_extents(&childExt, child_idx);
            walk_tree(tree, child_idx, &childExt, process_callback, callback_arg);
        }
    }
}

//endregion idx tree traversal

//region oct node operations

void setNodeEmpty(struct OctNode* node) {
    memset(node, 0, sizeof(struct OctNode));
}

void setNodeToLeafNode(struct OctNode* node) {
    node->contentType = CT_LEAVES;
    node->size = 0;
}

void addLeafToLeafNode(struct OctNode* node, leaf_idx_t idx) {
    dbgAssert(node->size < LEAF_CHILD_COUNT);
    node->leaves[node->size] = idx;
    node->size++;
}

void removeLeafFromNode(struct OctNode* node, child_pos_idx_t idx) {
    dbgAssert(node->size > 0);

    node->size--;
    if(node->size == 0) {
        setNodeEmpty(node);
        return;
    }

    for(child_pos_idx_t i = idx; i < node->size-1; i++) {
        node->leaves[i] = node->leaves[i+1];
    }
}

void scatterLeavesInNode(struct OctTree* tree, node_idx_t idx, struct Extents* ext) {
    dbgAssert(getNode(tree, idx)->contentType == CT_LEAVES);

    child_pos_idx_t sizeTarget = LEAF_CHILD_COUNT - MAX_SCATTER;

    while(true) {
        rebalance_node(tree, idx, ext, nullptr);

        if(getNode(tree, idx)->size <= sizeTarget) break;

        getLeaf(tree, idx, 0)->pos = rand_pos();
    }
}

bool setNodeToInternalNode(struct OctTree* tree, node_idx_t idx, struct Extents* ext) {

    if(tree->depth_count == DEPTH_LIMIT) {
//        scatterLeavesInNode(tree, idx, ext);
        return false;
    }

    node_idx_t size = getNode(tree, idx)->size;
    leaf_idx_t leaves[LEAF_CHILD_COUNT];
    for(leaf_idx_t i = 0; i < LEAF_CHILD_COUNT; i++)
        leaves[i] = getNode(tree, idx)->leaves[i];

    getNode(tree, idx)->contentType = CT_NODES;
    getNode(tree, idx)->size = 0;

    for(node_idx_t i = 0; i < size; i++) {
        struct Extents childExt = *ext;
        addLeafToNode(tree, leaves[i], idx, &childExt);
    }

    return true;
}

//endregion oct node operations

//region add leaf

struct Extents get_max_extents() {
    struct Extents ret;
    ret.maxExt.x =  UNIVERSE_SIZE;
    ret.maxExt.y =  UNIVERSE_SIZE;
    ret.maxExt.z =  UNIVERSE_SIZE;
    ret.minExt.x = -UNIVERSE_SIZE;
    ret.minExt.y = -UNIVERSE_SIZE;
    ret.minExt.z = -UNIVERSE_SIZE;
    ret.center.x =  0;
    ret.center.y =  0;
    ret.center.z =  0;
    return ret;
}

bool pos_inside(struct Extents* ext, Pos* pos) {
    return (
        (ext->minExt.x <= pos->x && pos->x < ext->maxExt.x) &&
        (ext->minExt.y <= pos->y && pos->y < ext->maxExt.y) &&
        (ext->minExt.z <= pos->z && pos->z < ext->maxExt.z)
    );
}

void update_extents(struct Extents* ext, child_pos_idx_t pos_index) {
    int xBit = 1 << 2;
    int yBit = 1 << 1;
    int zBit = 1;

    if((pos_index & xBit) > 0) {
        ext->minExt.x = ext->center.x;
    } else {
        ext->maxExt.x = ext->center.x;
    }

    if((pos_index & yBit) > 0) {
        ext->minExt.y = ext->center.y;
    } else {
        ext->maxExt.y = ext->center.y;
    }

    if((pos_index & zBit) > 0) {
        ext->minExt.z = ext->center.z;
    } else {
        ext->maxExt.z = ext->center.z;
    }

    ext->center.x = (ext->minExt.x + ext->maxExt.x) / 2;
    ext->center.y = (ext->minExt.y + ext->maxExt.y) / 2;
    ext->center.z = (ext->minExt.z + ext->maxExt.z) / 2;
}

child_pos_idx_t get_pos_index(struct Extents* ext, Pos* pos) {

    child_pos_idx_t ret = 0;

    if(ext->center.x <= pos->x) ret += 1;
    ret = ret << 1;

    if(ext->center.y <= pos->y) ret += 1;
    ret = ret << 1;

    if(ext->center.z <= pos->z) ret += 1;

    return ret;
}

bool addLeafToNode(struct OctTree* tree, leaf_idx_t leaf_idx, node_idx_t idx, struct Extents* ext) {

    depth_t idx_depth = get_depth_for_idx(idx);
    if(idx_depth >= tree->depth_count) {
        set_tree_depth(tree, idx_depth + 1);
    }

    Pos* leafPos = &tree->leaves[leaf_idx].pos;
//    dbgAssert(pos_inside(ext, leafPos));

    if(getNode(tree, idx)->contentType == CT_EMPTY) {
        setNodeToLeafNode(getNode(tree, idx));
    }

    if(getNode(tree, idx)->contentType == CT_LEAVES) {
        if(getNode(tree, idx)->size < LEAF_CHILD_COUNT) {
            addLeafToLeafNode(getNode(tree, idx), leaf_idx);
            return true;
        } else {
            if(!setNodeToInternalNode(tree, idx, ext)) {
                return false;
            }
//            if(getNode(tree, idx)->contentType != CT_NODES) {
//                // hit depth limit, node has scattered so add
//                if(getNode(tree, idx)->contentType == CT_EMPTY)
//                    setNodeToLeafNode(getNode(tree, idx));
//                addLeafToLeafNode(getNode(tree, idx), leaf_idx);
//                return;
//            }
        }
    }

    if(getNode(tree, idx)->contentType == CT_NODES) {
        child_pos_idx_t posIdx = get_pos_index(ext, leafPos);
        update_extents(ext, posIdx);
        if(addLeafToNode(tree, leaf_idx, get_node_children(idx) + posIdx, ext)) {
            getNode(tree, idx)->size++;
            return true;
        }

        return false;
    }

    dbgAssert(false);
}
//endregion add leaf

//region center of mass

void calc_node_center_of_mass(struct OctTree* tree, node_idx_t idx) {
    if(getNode(tree, idx)->contentType == CT_EMPTY) return;

    if(getNode(tree, idx)->contentType == CT_LEAVES) {
        getNode(tree, idx)->centerOfMass.x = 0;
        getNode(tree, idx)->centerOfMass.y = 0;
        getNode(tree, idx)->centerOfMass.z = 0;

        for(int i = 0; i < getNode(tree, idx)->size; i++) {
            struct Leaf* leaf = getLeaf(tree, idx, i);
            getNode(tree, idx)->centerOfMass.x += leaf->pos.x / (float) getNode(tree, idx)->size;
            getNode(tree, idx)->centerOfMass.y += leaf->pos.y / (float) getNode(tree, idx)->size;
            getNode(tree, idx)->centerOfMass.z += leaf->pos.z / (float) getNode(tree, idx)->size;
        }

        return;
    }

    if(getNode(tree, idx)->contentType == CT_NODES) {
        getNode(tree, idx)->centerOfMass.x = 0;
        getNode(tree, idx)->centerOfMass.y = 0;
        getNode(tree, idx)->centerOfMass.z = 0;

        node_idx_t children_start = get_node_children(idx);
        for(int i = 0; i < NODE_CHILD_COUNT; i++) {
            calc_node_center_of_mass(tree, children_start+i);

            getNode(tree, idx)->centerOfMass.x += (getNode(tree, children_start+i)->centerOfMass.x * (float) getNode(tree, children_start+i)->size) / (float) getNode(tree, idx)->size;
            getNode(tree, idx)->centerOfMass.y += (getNode(tree, children_start+i)->centerOfMass.y * (float) getNode(tree, children_start+i)->size) / (float) getNode(tree, idx)->size;
            getNode(tree, idx)->centerOfMass.z += (getNode(tree, children_start+i)->centerOfMass.z * (float) getNode(tree, children_start+i)->size) / (float) getNode(tree, idx)->size;
        }
        return;
    }

    dbgAssert(false);
}

//endregion center of mass

//region calc force on node
bool calc_force_for_node(struct OctTree* tree, node_idx_t idx, struct Extents* ext, void* callback_arg) {
    struct Leaf* leaf = (struct Leaf*) callback_arg;

    coord_t s = ext->maxExt.x - ext->minExt.x;
    s = s > 0 ? s : -s;

    coord_t d = dist(&getNode(tree, idx)->centerOfMass, &leaf->pos);

    if(s/d < SD_THRESHOLD) { // node is far enough away

        coord_t f = (G*(getNode(tree, idx)->size + 1))/(d*d);
        Force fVec = vec_dir(&leaf->pos, &getNode(tree, idx)->centerOfMass);
        mult_scalar(&fVec, f);

        add(&leaf->force, &fVec);

        return false;
    } else { // node is too close
        if(getNode(tree, idx)->contentType == CT_LEAVES) {
            for(leaf_idx_t i = 0; i < getNode(tree, idx)->size; i++) {
                struct Leaf* otherLeaf = getLeaf(tree, idx, i);
                coord_t d = dist(&leaf->pos, &otherLeaf->pos);
                if(d == 0) continue;
                coord_t f = (G*2)/(d*d);
                Force fVec = vec_dir(&leaf->pos, &otherLeaf->pos);
                mult_scalar(&fVec, f);

                add(&leaf->force, &fVec);
            }
        } else {
            return true;
        }
    }

    return true;
}

void calc_force_on_leaf(struct OctTree* tree, struct Leaf* leaf) {
    struct Extents rootExt = get_max_extents();
    walk_tree(tree, 0, &rootExt, calc_force_for_node, leaf);
}
//endregion calc force on node

//region apply force
// i dont know if this is right way to kernalize this function
__global__ void apply_force_on_leaf(struct OctTree tree, struct Leaf* leaf) {
    *leaf->force = blockDim.x * blockIdx.x + threadIdx.x;
    *leaf->velocity = blockDim.y * blockIdx.y + threadIdx.y;
    mult_scalar(&leaf->force, FORCE_MULT);
    add(&leaf->velocity, &leaf->force);
}

// void apply_force_on_leaf(struct OctTree* tree, struct Leaf* leaf) {
//     mult_scalar(&leaf->force, FORCE_MULT);
//     add(&leaf->velocity, &leaf->force);
// }
//endregion apply force

__global__ void apply_velocity_on_leaf(struct OctTree tree, struct Leaf* leaf) {
    *leaf->velocity = blockDim.x * blockIdx.x + threadIdx.x;
    *leaf->pos = blockDim.y * blockIdx.y + threadIdx.y;
    Velocity *v = leaf->velocity;
    mult_scalar(&v, VELOCITY_MULT);
    add(&leaf->pos, &v);
    clamp_to_universe(&leaf->pos, &leaf->velocity);
}
//region apply velocity
// void apply_velocity_on_leaf(struct OctTree* tree, struct Leaf* leaf) {
//     Velocity v = leaf->velocity;
//     mult_scalar(&v, VELOCITY_MULT);
//     add(&leaf->pos, &v);
//     clamp_to_universe(&leaf->pos, &leaf->velocity);
// }
//endregion apply velocity

//region rebalance
bool rebalance_node(struct OctTree* tree, node_idx_t idx, struct Extents* ext, void* callback_arg) {
    for(node_idx_t i = getNode(tree, idx)->size-1; getNode(tree, idx)->contentType == CT_LEAVES; i--) {
        struct Leaf* leaf = getLeaf(tree, idx, i);
        if(!pos_inside(ext, &leaf->pos)) {
            leaf_idx_t leafIdx = getNode(tree, idx)->leaves[i];

            removeLeafFromNode(getNode(tree, idx), i);

            node_idx_t parentIdx = get_node_parent(idx);
            while(true) {
                getNode(tree, parentIdx)->size--;
                if(getNode(tree, parentIdx)->size == 0) setNodeEmpty(getNode(tree, parentIdx));
                if(parentIdx == 0) break;
                parentIdx = get_node_parent(parentIdx);
            }

            addLeaf(tree, leafIdx);
        }
        if(i == 0) break;
    }

    return true;
}
//endregion rebalance

////// PUBLIC SECTION ////

// within these, allocate and destroy GPU memory
struct OctTree* create_tree(leaf_idx_t leaf_count) {
    CUDA_SAFE_CALL(hipSetDevice(0));
    leaf_idx_t allocSizeTree = sizeof(struct OctTree);
    leaf_idx_t allocSizeLeaf = sizeof(struct Leaf);
    struct OctTree* tree;
    
    // struct OctTree* tree = (struct OctTree*) malloc(sizeof(struct OctTree));
    CUDA_SAFE_CALL(hipMalloc((void **)&tree, allocSizeTree));

    // tree->leaves = calloc(leaf_count, sizeof(struct Leaf));
    // memset(tree->leaves, 0, leaf_count*sizeof(struct Leaf));
    CUDA_SAFE_CALL(hipMalloc((void **)&tree->leaves, leaf_count * allocSizeLeaf));

    tree->leaf_count = leaf_count;

    tree->depth = nullptr;
    tree->depth_count = 0;
    set_tree_depth(tree, 1);

    return tree;
}

void destroy_tree(struct OctTree* tree) {
    // free(tree->leaves);
    CUDA_SAFE_CALL(hipFree(tree->leaves));
    
    for(depth_t i = 0; i < tree->depth_count; i++) {
        free(tree->depth[i]);
    }

    free(tree->depth);

    CUDA_SAFE_CALL(hipFree(tree->depth));
    
    // free(tree);
    CUDA_SAFE_CALL(hipFree(tree));
}

void add_leaves_to_tree(struct OctTree* tree) {
    for(leaf_idx_t i = 0; i < tree->leaf_count; i++) {
        addLeaf(tree, i);
    }
}

void addLeaf(struct OctTree* tree, leaf_idx_t leaf_idx) {
    while(true) {
        struct Extents ext = get_max_extents();
        if(addLeafToNode(tree, leaf_idx, 0, &ext)) {
            break;
        }
        tree->leaves[leaf_idx].pos = rand_pos();
    }
}

void calc_center_of_mass(struct OctTree* tree) {
    calc_node_center_of_mass(tree, 0);
}

__global__ void walk_leaves_global(struct OctTree* tree, void (*process_leaf)(struct OctTree* tree, struct Leaf* leaf)) {
    leaf_idx_t i_start = blockDim.x * blockIdx.x + threadIdx.x;
    process_leaf(tree, &tree->leaves[i_start]);
}

void calc_force(struct OctTree* tree) {
    // walk_leaves(tree, calc_force_on_leaf);

    // Launch kernel
    dim3 dimGrid(64, 64);
    dim3 dimBlock(16, 16);
    walk_leaves_global<<<dimGrid, dimBlock>>>(tree, calc_force_on_leaf);

    // Check for launch errors
    CUDA_SAFE_CALL(hipPeekAtLastError());
}

void apply_force(struct OctTree* tree) {
    // walk_leaves(tree, apply_force_on_leaf);

    // apply force on leaf is kernel. must change pointers to take leaf array. calculate index. copy vector functions into device functions
    // separate function into GPU, do the functions, and recopy memory back into host
    
    // Launch kernel
    dim3 dimGrid(64, 64);
    dim3 dimBlock(16, 16);
    walk_leaves_global<<<dimGrid, dimBlock>>>(tree, apply_force_on_leaf<<<dimGrid, dimBlock>>>(tree, tree->leaves)); // copy memory into GPU
    
    // Check for launch errors
    CUDA_SAFE_CALL(hipPeekAtLastError());
}

void apply_velocity(struct OctTree* tree) {
    // walk_leaves(tree, apply_velocity_on_leaf);
    
    // Launch kernel
    dim3 dimGrid(64, 64);
    dim3 dimBlock(16, 16);
    walk_leaves_global<<<dimGrid, dimBlock>>>(tree, apply_velocity_on_leaf<<<dimGrid, dimBlock>>>(tree, tree->leaves)); // pass GPU memory leaves directly
    
    // Check for launch errors
    CUDA_SAFE_CALL(hipPeekAtLastError());
}

void rebalance(struct OctTree* tree) {
    struct Extents ext = get_max_extents();
    walk_tree(tree, 0, &ext, rebalance_node, nullptr);
}
