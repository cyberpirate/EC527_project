#include "hip/hip_runtime.h"
//
// Created by cyberpirate on 3/30/22.
//

#include "rand_gen.cuh"
#include "oct_tree.cuh"
#include <stdlib.h>
#include <math.h>

#define PI 3.14

void reset_rand() {
    srand(RAND_SEED);
}

coord_t rand_value(coord_t limit) {
    coord_t ret = (coord_t) rand();
    ret /= (float) (RAND_MAX/2);
    ret -= 1;
    ret *= limit;
    return ret;
}

coord_t rand_coord() {
    return rand_value(UNIVERSE_SIZE);
}

Pos rand_pos() {
    Pos ret;
    ret.x = rand_coord();
    ret.y = rand_coord();
    ret.z = rand_coord();
    return ret;
}

Pos rand_torus(coord_t outer_radius, coord_t inner_radius) {
    Pos ret;

    coord_t diff = rand_value(inner_radius);
    coord_t r = outer_radius + diff;
    coord_t theta = rand_value(PI) + PI;

    ret.x = r*cos(theta);
    ret.y = r*sin(theta);
    ret.z = rand_value(inner_radius*cos((diff/inner_radius)*(PI/2)));

    return ret;
}

Velocity vel_from_pos(Pos* pos) {
    Velocity ret;
    ret.x = -pos->y;
    ret.y = pos->x;
    ret.z = 0;
    return ret;
}
